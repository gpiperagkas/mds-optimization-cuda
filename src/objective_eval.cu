
// Copyright 2025 Grigorios Piperagkas. All rights reserved.
// Use of this source code is governed by a BSD-3-clause
// license that can be found in the LICENSE file.
/*
//////////////////////////////////////////////////////////////////////////////

Main algorithm for Parallel Multidirectional Search proposed by VJ Torczon(1989).
implemented for evaluation of objective function in parallel on cuda enabled GPUs.

June 2025.
/////////////////////////////////////////////////////////////////////////////

 */


#include <hip/hip_runtime.h>
#include <math.h>

__global__ void evaluate_simplex_sphere(float *a, float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        b[index] = 0.0;
        for (int i=0;i<n-1;i++)
            b[index] = b[index] + a[index*(n-1)+i]*a[index*(n-1)+i];
    }
}

__global__ void evaluate_simplex_rosenbrock(float *a, float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        b[index] = 0.0;
        for (int i=0;i<n-2;i++)
            b[index] = b[index] + 100*(a[index*(n-1)+i+1] - a[index*(n-1)+i]*a[index*(n-1)+i])*(a[index*(n-1)+i+1] - a[index*(n-1)+i]*a[index*(n-1)+i]) +
                    (a[index*(n-1)+i]-1)*(a[index*(n-1)+i]-1);
    }
}

__global__ void evaluate_simplex_rastrigin(float *a, float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    double pi=3.14159265358979;
     if (index < n) {
            b[index] = 0.0;
            for (int i=0;i<n-1;i++)
                b[index] = b[index] + a[index*(n-1)+i]*a[index*(n-1)+i] + 10 -10*cosf(2*pi*a[index*(n-1)+i]);
        }
}


extern "C" void objective(float *simplex, float *fsimplex, int n, int benchmark) {
    int threads_per_block = 1024;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    int solution[n-1];
    if (benchmark==0){
        evaluate_simplex_sphere<<<blocks_per_grid, threads_per_block>>>(simplex, fsimplex, n);
        for (int j=0;j<n-1;j++)
            solution[j]=0;
    }else if (benchmark==1){
        evaluate_simplex_rosenbrock<<<blocks_per_grid, threads_per_block>>>(simplex, fsimplex, n);
    }else if (benchmark==2){
        evaluate_simplex_rastrigin<<<blocks_per_grid, threads_per_block>>>(simplex, fsimplex, n);
    }

    hipDeviceSynchronize();
}

